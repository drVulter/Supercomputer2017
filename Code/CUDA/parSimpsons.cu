#include "hip/hip_runtime.h"
/*
Parallel implementation of Simpson's rule in CUDA
(annoyed grunt)
*/

#include<stdio.h>

__device__
float myFunc(float x)
{
	return x*x;
}
// kernel Function
__global__
void kernelFunc(float a, float width, int N, float *valOut) {
	int i = threadIdx.x + 1; // Begin at zero so 
	float x = a + ((float)i * width);
	if (i < N) {
		if (i % 2 == 1) {
			valOut[i] = 4 * myFunc(x);
		} else {
			valOut[i] = 2 * myFunc(x);
		}
	}
}
int main() {

 	float a, b; // Endpoints
	int N = 1000; // Number of subintervals
	int i; // counter
	float sum; // Holds sum
	float width; // width of subintervals
	float result = 0; // final result

	// array to carry values out of kernel
	float *valOut = 0;
	// Allocate device memory for output array
	hipMalloc(&valOut, N*sizeof(float));

	// Array for host
	float hostArray* = (float*)calloc(N, sizeof(float));

	a = 0.0;
	b = 100.0;
	width = (b - a) / N;

	kernelFunc<<<1, 256>>>(a, width, N, valOut);

	hipMemcpy(hostArray, valOut, N*sizeof(float), hipMemcpyDeviceToHost);

	for (i = 0; i < N; i++) {
		sum += hostArray[i];
	}

	result = (sum * width) / 3.0;

	printf("The value is %f\n", result);

	// Clean up
	free(hostArray);
	hipFree(valOut);

	return 0;

}
